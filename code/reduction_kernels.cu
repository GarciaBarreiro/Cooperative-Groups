
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <sys/time.h>

#ifndef tileSize
    #define tileSize 16
#else
    #if tileSize < 1 || tileSize > 32
        #error "tileSize debe ser un valor entre 1 e 32"
    #endif
#endif

#ifndef iters
    #define iters 1000
#endif

#define INIT_TIME(prev, init) \
    gettimeofday(&prev, NULL); \
    gettimeofday(&init, NULL);

// remove overhead created by call to gettimeofday
#define GET_TIME(prev, init, final, res) \
    gettimeofday(&final, NULL); \
    res = (final.tv_sec-init.tv_sec+(final.tv_usec-init.tv_usec)/1.e6) - \
          (init.tv_sec-prev.tv_sec+(init.tv_usec-prev.tv_usec)/1.e6);

/*
 Calcula a suma de val. O array x, temporal e en memoria distribuída,
 ten que ser o suficientemente grande para conter blockDim.x enteiros. O resultado
 esperado será (n-1)*n/2, tendo en conta que o primeiro fío ten rango 0
*/
__device__ int sumReduction(int *x, int val) {
    // rango do fío dentro do bloque
    int lane = threadIdx.x;

    // redución, de tal xeito que o resultado quede en val no fío con rango 0
    for (int i = blockDim.x / 2; i > 0; i /= 2) {
        x[lane] = val;
        __syncthreads();
        if (lane < i) { val += x[lane + i]; }
        __syncthreads();
    }

    // o fío con rango 0 devolve o resultado, o resto -1
    if (lane == 0) { return val; }
    else { return -1; }
}

// Kernel, crea grupos cooperativos e realiza reducións
__global__ void cgkernel() {
    // array temporal para a redución
    extern __shared__ int workspace[];

    int input, output, expectedOutput;
    input = threadIdx.x;

    // resultado esperado, usando a fórmula previamente mencionada
    expectedOutput = (blockDim.x - 1) * blockDim.x / 2;
    
    output = sumReduction(workspace, input);

    // o fío mestre imprime o resultado
    if (threadIdx.x == 0) {
        printf("Suma de 0 a %d no bloque %d é %d, esperado %d\n",
            blockDim.x - 1, blockIdx.x, output, expectedOutput);
    }

    return;
}

int main() {
    struct timeval init, prev, final;
    double time;

    INIT_TIME(prev, init);
    for (int _ = 0; _ < iters; _++) {
        int blocksPerGrid = 1;
        int threadsPerBlock = 1024;

        printf("Usando %d bloques de %d fíos\n", blocksPerGrid, threadsPerBlock);

        cgkernel<<<blocksPerGrid, threadsPerBlock, threadsPerBlock * sizeof(int)>>>();
        hipError_t err = hipDeviceSynchronize();
        if (err != hipSuccess) {
            fprintf(stderr, "Erro de CUDA: %s\n", hipGetErrorString(err));
            return 1;
        }

        threadsPerBlock = tileSize;
        blocksPerGrid = 1024 / tileSize;

        printf("Usando %d bloques de %d fíos\n", blocksPerGrid, threadsPerBlock);

        cgkernel<<<blocksPerGrid, threadsPerBlock, threadsPerBlock * sizeof(int)>>>();
        err = hipDeviceSynchronize();
        if (err != hipSuccess) {
            fprintf(stderr, "Erro de CUDA: %s\n", hipGetErrorString(err));
            return 1;
        }
    }
    GET_TIME(prev, init, final, time);
    printf("Tempo total: %f s\n", time);

    FILE *fp = fopen("kernels.csv", "a");
    fprintf(fp, "%d,%d,%f\n", tileSize, iters, time);
    fclose(fp);

    return 0;
}
